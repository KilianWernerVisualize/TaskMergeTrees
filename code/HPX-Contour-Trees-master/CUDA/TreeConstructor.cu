#include "hip/hip_runtime.h"
#ifndef TREECONSTRUCTOR_CU
#define TREECONSTRUCTOR_CU
#include "TreeConstructor.h"
#include <chrono>
#include <stack>
//#include <hip/hip_cooperative_groups.h>

#include <stdexcept>

__device__ int minimaQueueCounter;
__device__ int doneCounter;
__device__ int minimaCounter;
__device__ double min_value;
__device__ double conversionFactor;

__device__ int minimarounds;
__device__ int growthrounds;

__device__ int max_x;
__device__ int max_y;
__device__ int max_z;

__device__ int boundarySize;
__device__ int* boundaryStart;

__device__ int trunkStart;

// -------------------------------------------------------------------------

/// throw exception if a CUDA error was generated in the wrapped call
inline void cuda_check( hipError_t code )
{
    if( code != hipSuccess )
        throw std::runtime_error( std::string( "CUDA error: " ) + hipGetErrorString(code) );
}

/// throw exception if a CUDA error occurred in the past
inline void cuda_check()
{
    cuda_check( hipGetLastError() );
}


// -------------------------------------------------------------------------

/// a simple wrapper for cudaArray3D
template<typename T>
struct array3D
{
    dim3 size() const;

    /// resize to (nx, ny, nz); array will be uninitialized
    void resize( dim3 size );

    /// copy host data (nx*ny*nz) elements into array
    void copy( const T* );

    /// return result of an texture lookup (interpolated read)
    __device__ T get(int idx) const;

    hipArray*          m_array = 0;
    hipExtent          m_extent;
    hipTextureObject_t m_texture;
};

// -------------------------------------------------------------------------

template<typename T>
dim3 array3D<T>::size() const
{
    return dim3( m_extent.width, m_extent.height, m_extent.depth );
}
// -------------------------------------------------------------------------

template<typename T>
void array3D<T>::resize( dim3 size )
{
    if( m_array )
    {
        // free a poossibly previously allocated array
        // and the associated texture object
        hipDestroyTextureObject( m_texture );
        hipFreeArray( m_array );

        m_array = 0;
    }

    m_extent = make_hipExtent( size.x, size.y, size.z );

    auto cdesc = hipCreateChannelDesc<T>();

    cuda_check( hipMalloc3DArray(
        &m_array,
        &cdesc,
        make_hipExtent(
            m_extent.width*sizeof(T),
            m_extent.height,
            m_extent.depth
        ), 0 )
    );

    // set up texture
    hipResourceDesc tr;
    memset( &tr, 0, sizeof(hipResourceDesc) );
    tr.resType         = hipResourceTypeArray;
    tr.res.array.array = m_array;

    hipTextureDesc td;
    memset( &td, 0, sizeof(hipTextureDesc) );

    td.filterMode       = hipFilterModeLinear;
    td.addressMode[0]   = hipAddressModeBorder;
    td.addressMode[1]   = hipAddressModeBorder;
    td.addressMode[2]   = hipAddressModeBorder;
    td.readMode         = hipReadModeElementType;
    td.normalizedCoords = false;

    cuda_check( hipCreateTextureObject(
        &m_texture,
        &tr,
        &td,
        NULL
    ) );
}

// -------------------------------------------------------------------------

template<typename T>
void array3D<T>::copy( T const *data  )
{
    hipMemcpy3DParms copyParams = { 0 };
    copyParams.srcPtr = make_hipPitchedPtr(
        (void*)data,
        m_extent.width*sizeof(T),
        m_extent.width,
        m_extent.height
    );

    copyParams.dstArray = m_array;
    copyParams.extent   = m_extent;
    copyParams.kind     = hipMemcpyHostToDevice;

    cuda_check( hipMemcpy3D( &copyParams ) );
}

// -------------------------------------------------------------------------

template<typename T>
__device__ T array3D<T>::get( int idx ) const
{
    return tex3D<T>( m_texture, idx % max_x, ( idx / max_x ) % max_y, ( idx / ( max_x*max_y )));
}

__global__ void min_search_preNeigh(int numVertices, array3D<float> values, int* neighborsBuffer, NeighborsEntry* neighborsMap, int* minimaQueue, int* swept, int* augmentation, unsigned long long int* saddleMap){
    int idx = blockIdx.x*blockDim.x + threadIdx.x;

    NeighborsEntry myNeigh;

    bool isMini;
    double neighVal;
    double myVal;
    int j;

    for(; idx<numVertices; idx+=gridDim.x*blockDim.x){
        swept[idx] = -1;
        augmentation[idx] = -1;
        saddleMap[idx] = 0-1;
        isMini = true;
        myNeigh = neighborsMap[idx];
        myVal = values.get(idx);
        for (j = 0; (j < myNeigh.numNeighbors); j++){
            neighVal = values.get(neighborsBuffer[myNeigh.offset+j]);
            if (neighVal < myVal || ((neighVal == myVal) && (neighborsBuffer[myNeigh.offset+j] < idx))){
                isMini = false;
            }
        }
        if (isMini){
            j = atomicAdd(&minimaQueueCounter, 1);
            minimaQueue[j] = idx;
        }
    }
}

__global__ void min_growth_preNeigh(int numVertices, array3D<float> values, int* neighborsBuffer, NeighborsEntry* neighborsMap, int* minimaQueue, int* swept, unsigned long long int* saddleMap, int** sweepQueueStarters){
    int idx = blockIdx.x*blockDim.x + threadIdx.x;

    int v;
    int touched;
    bool sweepable;

    double neighVal;
    double myVal;
    int j;
    int i;

    int stackCount = -1;
    int stack[1000];

    NeighborsEntry myNeigh;

    for (; idx < minimaQueueCounter; idx+=gridDim.x*blockDim.x){

        v = minimaQueue[idx];
        myNeigh = neighborsMap[v];
        stackCount = -1;
        for (j = 0; (j < myNeigh.numNeighbors); j++){
            stack[stackCount + j +1] = neighborsBuffer[myNeigh.offset + j];
        }
        stackCount += myNeigh.numNeighbors;
        swept[v] = v;

        i = 0;
        while (stackCount >= 0 && (i < minimarounds)){
            i++;
            sweepable = true;
            touched = stack[stackCount];
            myVal = values.get(touched);
            stackCount--;

            myNeigh = neighborsMap[touched];
            for (j = 0; (j < myNeigh.numNeighbors); j++){
                neighVal = values.get(neighborsBuffer[myNeigh.offset + j]);
                if ((neighVal < myVal || ((neighVal == myVal) && (neighborsBuffer[myNeigh.offset+j] < touched))) && ((swept[neighborsBuffer[myNeigh.offset + j]] >= -1) && (swept[neighborsBuffer[myNeigh.offset + j]] != v))){
                    sweepable = false;
                }
            }
            if (sweepable){
                swept[touched] = v;
                for (j = 0; (j < myNeigh.numNeighbors); j++){
                    if ((swept[neighborsBuffer[myNeigh.offset + j]] >= -1) && (swept[neighborsBuffer[myNeigh.offset + j]] != v)){
                        stackCount++;
                        if (stackCount >= 1000){
                            printf("Oh no!");
                            stackCount = -1;
                        } else {
                            stack[stackCount] = neighborsBuffer[myNeigh.offset + j];
                        }
                    }
                }
            }
        }
        if (stackCount >= 0){
            sweepQueueStarters[idx + (int)(numVertices/2)] = (int*)malloc((stackCount+1)*sizeof(int));
            for (i = 0; (i <= stackCount); i++){
                sweepQueueStarters[idx + (int)(numVertices/2)][i] = stack[i];
            }
            saddleMap[v] = (((unsigned long long int)idx) << 32) + stackCount+1 + 1;
            swept[v] = -2 - idx;
        } else {
            sweepQueueStarters[idx] = nullptr;
            swept[v] = v;
        }
    }
}

__global__ void saddle_growth(int numVertices, array3D<float> values, int* neighborsBuffer, NeighborsEntry* neighborsMap, int* minimaQueue, int* swept, unsigned long long int* saddleMap, int** sweepQueueStarters, int round){
    int idx = blockIdx.x*blockDim.x + threadIdx.x;

    int v;
    int touched;
    bool sweepable;

    double neighVal;
    double myVal;
    int j;
    int i;

    int stackCount = -1;
    int stack[1000];

    NeighborsEntry myNeigh;

    for (; idx < minimaQueueCounter; idx+=gridDim.x*blockDim.x){
        v = minimaQueue[idx];
        myNeigh = neighborsMap[v];
        stackCount = -1;
        for (j = 0; (j < myNeigh.numNeighbors); j++){
            stack[stackCount + j +1] = neighborsBuffer[myNeigh.offset + j];
        }
        stackCount += myNeigh.numNeighbors;

        swept[v] = v;

        i = 0;
        while ((stackCount >= 0 || ((unsigned int)saddleMap[v]) > 1) && (i < growthrounds)){
            i++;
            sweepable = true;
            if (stackCount >= 0){
                touched = stack[stackCount];
                stackCount--;
            } else {
                saddleMap[v]--;
                touched = sweepQueueStarters[idx][((unsigned int)saddleMap[v])-1];
                //printf("Taking %d from %u at %u \n", touched, v, (((unsigned int)saddleMap[v])-1));
            }
            myVal = values.get(touched);

            myNeigh = neighborsMap[touched];
            for (j = 0; (j < myNeigh.numNeighbors); j++){
                neighVal = values.get(neighborsBuffer[myNeigh.offset + j]);
                if ((neighVal < myVal || ((neighVal == myVal) && (neighborsBuffer[myNeigh.offset+j] < touched))) && ((swept[neighborsBuffer[myNeigh.offset + j]] >= -1) && (swept[neighborsBuffer[myNeigh.offset + j]] != v))){
                    sweepable = false;
                }
            }
            if (sweepable){
                swept[touched] = v;
                for (j = 0; (j < myNeigh.numNeighbors); j++){
                    if ((swept[neighborsBuffer[myNeigh.offset + j]] >= -1) && (swept[neighborsBuffer[myNeigh.offset + j]] != v)){
                        stackCount++;
                        if (stackCount >= 1000){
                            printf("Oh no!");
                            stackCount = -1;
                        } else {
                            stack[stackCount] = neighborsBuffer[myNeigh.offset + j];
                        }
                    }
                }
            }
        }
        if (stackCount >= 0 || ((unsigned int)saddleMap[v]) > 1){
            j = (stackCount+((unsigned int)saddleMap[v]));
            int* tmp = (int*)malloc(j*sizeof(int));
            for (i = 0; (i <= stackCount); i++)
                tmp[i] = stack[i];
            for (i = stackCount+1; ( i < j); i++){
                tmp[i] = sweepQueueStarters[idx][i-stackCount-1];
            }
            free(sweepQueueStarters[idx]);
            sweepQueueStarters[idx + (int)(numVertices/2)] = tmp;
            saddleMap[v] = (((unsigned long long int)idx) << 32) + j + 1;
            swept[v] = -2 - idx;
        } else {
            free(sweepQueueStarters[idx]);
            sweepQueueStarters[idx] = nullptr;
            swept[v] = v;
        }
    }
}

__global__ void clean_saddleMap_to_infinity(int numVertices, int* swept, unsigned long long int* saddleMap){
    int idx = blockIdx.x*blockDim.x + threadIdx.x;

    for(; idx<numVertices; idx+=gridDim.x*blockDim.x){
        if (swept[idx] >= -1)
            saddleMap[idx] = 0-1;
    }
}

__global__ void clean_saddleMap_to_zero(int numVertices, int* swept, unsigned long long int* saddleMap){
    int idx = blockIdx.x*blockDim.x + threadIdx.x;

    for(; idx<numVertices; idx+=gridDim.x*blockDim.x){
        if (saddleMap[idx] != 0-2 && swept[idx] >= -1)
            saddleMap[idx] = 0;
    }
}

__global__ void saddle_search(int numVertices, array3D<float> values, int* neighborsBuffer, NeighborsEntry* neighborsMap, int* minimaQueue, int* swept, unsigned long long int* saddleMap){
    int idx = blockIdx.x*blockDim.x + threadIdx.x;

    NeighborsEntry myNeigh;
    int j;
    double neighVal;
    double myVal;
    unsigned int convertedValue;

    for(; idx<numVertices; idx+=gridDim.x*blockDim.x){
        if (swept[idx] == -1){
            myNeigh = neighborsMap[idx];
            myVal = values.get(idx);
            for (j = 0; (j < myNeigh.numNeighbors); j++){
                neighVal = values.get(neighborsBuffer[myNeigh.offset+j]);
                if (neighVal < myVal || ((neighVal == myVal) && (neighborsBuffer[myNeigh.offset+j] < idx))){
                    if ((swept[neighborsBuffer[myNeigh.offset+j]] >= 0) && (swept[swept[neighborsBuffer[myNeigh.offset+j]]] >= -1)){
                        convertedValue = static_cast<unsigned int>((myVal - min_value)*conversionFactor);
                        atomicMin(&saddleMap[swept[neighborsBuffer[myNeigh.offset+j]]], (((unsigned long long int)convertedValue) << 32) + static_cast<unsigned long long int>(idx));
                    }
                }
            }
        }
    }
}

__global__ void compress_saddles(int numVertices, array3D<float> values, int* neighborsBuffer, NeighborsEntry* neighborsMap, int* minimaQueue, int* swept, unsigned long long int* saddleMap, int** sweepQueueStarters){
    int idx = blockIdx.x*blockDim.x + threadIdx.x;

    NeighborsEntry myNeigh;
    int j;
    double neighVal;
    double myVal;
    bool saddle;
    bool checksaddle;


    for(; idx<numVertices; idx+=gridDim.x*blockDim.x){
        if (swept[idx] == -1){
            saddle = false;
            checksaddle = true;
            myNeigh = neighborsMap[idx];
            myVal = values.get(idx);

            for (j = 0; (j < myNeigh.numNeighbors); j++){
                neighVal = values.get(neighborsBuffer[myNeigh.offset+j]);
                if (neighVal < myVal || ((neighVal == myVal) && (neighborsBuffer[myNeigh.offset+j] < idx))){
                    if ((swept[neighborsBuffer[myNeigh.offset+j]] >= 0) && (swept[swept[neighborsBuffer[myNeigh.offset+j]]] >= -1)){
                        if (((unsigned int)saddleMap[swept[neighborsBuffer[myNeigh.offset+j]]]) == idx){
                            //if (swept[neighborsBuffer[myNeigh.offset+j]] != idx)
                            //swept[swept[neighborsBuffer[myNeigh.offset+j]]] = idx;
                            saddle = true;
                        } else {
                            checksaddle = false;
                        }
                    } else {
                        checksaddle = false;
                    }
                }
            }
            if (!checksaddle)
                saddleMap[idx] = 0-2;
            if (saddle && checksaddle){
                j = atomicAdd(&minimaQueueCounter, 1);
                minimaQueue[j] = idx;
            }
        } else if (swept[idx] <= -2) {
            j = atomicAdd(&minimaQueueCounter, 1);
            minimaQueue[j] = idx;
            sweepQueueStarters[j] = sweepQueueStarters[-2-swept[idx] + (int)(numVertices/2)];
            sweepQueueStarters[-2-swept[idx] + (int)(numVertices/2)] = nullptr;
        }
    }
}

__global__ void size_sweepqueues(int numVertices, array3D<float> values, int* neighborsBuffer, NeighborsEntry* neighborsMap, int* minimaQueue, int* swept, unsigned long long int* saddleMap, int** sweepQueueStarters){
    int idx = blockIdx.x*blockDim.x + threadIdx.x;

    NeighborsEntry myNeigh;
    int j;
    int i;
    double neighVal;
    double myVal;
    int mySaddle;
    int myMin[12];


    for(; idx<numVertices; idx+=gridDim.x*blockDim.x){
        if (swept[idx] == -1){
            mySaddle = -1;
            myNeigh = neighborsMap[idx];
            myVal = values.get(idx);

            for (j = 0; (j < myNeigh.numNeighbors); j++){
                myMin[j] = -1;
                neighVal = values.get(neighborsBuffer[myNeigh.offset+j]);
                if (neighVal < myVal || ((neighVal == myVal) && (neighborsBuffer[myNeigh.offset+j] < idx))){
                    myMin[j] = swept[neighborsBuffer[myNeigh.offset+j]];
                }
            }
            for (j = 0; (j < myNeigh.numNeighbors-1); j++){
                for (i = j+1; (i < myNeigh.numNeighbors); i++){
                    if (myMin[i] >= 0 && myMin[j] >= 0 && myMin[i] != myMin[j] && swept[myMin[i]] == swept[myMin[j]])
                        mySaddle = swept[myMin[i]];
                }
            }
            if (mySaddle != -1 && saddleMap[mySaddle] != 0-2)
                atomicAdd(&saddleMap[mySaddle], 1);
        }
    }
}

__global__ void create_sweepqueues(int numVertices, array3D<float> values, int* neighborsBuffer, NeighborsEntry* neighborsMap, int* minimaQueue, int* swept, unsigned long long int* saddleMap, int** sweepQueueStarters){
    int idx = blockIdx.x*blockDim.x + threadIdx.x;

    for(; idx<minimaQueueCounter; idx+=gridDim.x*blockDim.x){
        //printf("Idx: %d, MinimaQueue[idx]: %d \n", idx, minimaQueue[idx]);
        //if (saddleMap[minimaQueue[idx]] > 0)
            //printf("Idx: %d; sweepqueuestartersize: %llu \n",minimaQueue[idx], saddleMap[minimaQueue[idx]]);
        if (swept[minimaQueue[idx]] >= -1){
            sweepQueueStarters[idx] = (int*)malloc(saddleMap[minimaQueue[idx]]*sizeof(int));
            saddleMap[minimaQueue[idx]] = (((unsigned long long int)idx) << 32) + 1;
        }
    }
}

__global__ void fill_sweepqueues(int numVertices, array3D<float> values, int* neighborsBuffer, NeighborsEntry* neighborsMap, int* minimaQueue, int* swept, unsigned long long int* saddleMap, int** sweepQueueStarters){
    int idx = blockIdx.x*blockDim.x + threadIdx.x;

    NeighborsEntry myNeigh;
    int j;
    int i;
    double neighVal;
    double myVal;
    int myMin[12];
    int mySaddle;
    unsigned long long int pos;


    for(; idx<numVertices; idx+=gridDim.x*blockDim.x){
        if (swept[idx] == -1){
            mySaddle = -1;
            myNeigh = neighborsMap[idx];
            myVal = values.get(idx);

            for (j = 0; (j < myNeigh.numNeighbors); j++){
                myMin[j] = -1;
                neighVal = values.get(neighborsBuffer[myNeigh.offset+j]);
                if (neighVal < myVal || ((neighVal == myVal) && (neighborsBuffer[myNeigh.offset+j] < idx))){
                    myMin[j] = swept[neighborsBuffer[myNeigh.offset+j]];
                }
            }
            for (j = 0; (j < myNeigh.numNeighbors-1); j++){
                for (i = j+1; (i < myNeigh.numNeighbors); i++){
                    if (myMin[i] >= 0 && myMin[j] >= 0 && myMin[i] != myMin[j] && swept[myMin[i]] == swept[myMin[j]])
                        mySaddle = swept[myMin[i]];
                }
            }
            if (mySaddle != -1 && saddleMap[mySaddle] != 0-2){
                pos = atomicAdd(&saddleMap[mySaddle], 1);
                //printf("Adding %d to %u at %u \n", idx, minimaQueue[(unsigned int)(pos >> 32)], ((unsigned int)pos) -1);
                sweepQueueStarters[(unsigned int)(pos >> 32)][((unsigned int)pos) -1] = idx;
            }
        }
    }
}

__global__ void compress_UF(int numVertices, array3D<float> values, int* swept, int* augmentation){
    int idx = blockIdx.x*blockDim.x + threadIdx.x;

    for(; idx<numVertices; idx+=gridDim.x*blockDim.x){
        if (swept[idx] >= 0){
            if (swept[swept[idx]] >= 0){
                if (augmentation[idx] == -1){
                    if ((values.get(idx) < values.get(swept[swept[idx]])) || ((values.get(idx) == values.get(swept[swept[idx]])) && (idx < swept[swept[idx]])) ){
                        augmentation[idx] = swept[idx];
                    }
                }
                swept[idx] = swept[swept[idx]];
            }
        }
    }
}

__global__ void get_saddles(int numVertices, array3D<float> values, int* neighborsBuffer, NeighborsEntry* neighborsMap, int* minimaQueue, int* swept, unsigned long long int* saddleMap){
    int idx = blockIdx.x*blockDim.x + threadIdx.x;

    unsigned long long int saddle;

    for (; idx < minimaCounter; idx+=gridDim.x*blockDim.x){
       if (swept[minimaQueue[idx]] < -1){
           minimaQueue[idx] = minimaQueue[idx];
       } else {
           saddle = saddleMap[minimaQueue[idx]];
           swept[minimaQueue[idx]] = saddle;
           minimaQueue[idx] = saddle;
       }
    }

}

__global__ void flatten_swept(int numVertices, array3D<float> values, int* swept){
    int idx = blockIdx.x*blockDim.x + threadIdx.x;

    for(; idx<numVertices; idx+=gridDim.x*blockDim.x){
        if (swept[idx] >= 0){
            if (values.get(swept[idx]) < values.get(idx) || ((values.get(swept[idx]) == values.get(idx)) && (swept[idx] < idx))){
                swept[idx] = -1;
            }
        }
    }
}

__global__ void trunk_augment(int numVertices, int dangcount, array3D<float> values, int* swept, int* dangling){
    int idx = blockIdx.x*blockDim.x + threadIdx.x;

    for(; idx<numVertices; idx+=gridDim.x*blockDim.x){
        if (swept[idx] == -1){
            if (values.get(idx) < values.get(trunkStart)){
               swept[idx] = trunkStart;
            } else if (values.get(idx) > values.get(dangling[dangcount -1])){
                swept[idx] = dangling[dangcount-1];
            } else {
                int min = 0;
                int max = dangcount-1;
                while (true){
                    int j = (max+min)/2;
                    if (values.get(idx) > values.get(dangling[j])){
                        if (values.get(idx) < values.get(dangling[j+1])){
                            swept[idx] = dangling[j];
                            break;
                        } else {
                            min = j+1;
                            if (min > max){
                                swept[idx] = dangling[min];
                                break;
                            }
                        }
                    } else {
                        max = j-1;
                        if (min > max){
                            swept[idx] = dangling[min];
                            break;
                        }
                    }
                }
            }
        }
    }
}

    array3D<float> valuest;

void augmentTrunk(TreeConstructor &tree, int trunkStarter, std::vector<int> &danglingvect, int *&swept){

    int* sweptd;
    hipMalloc(&sweptd, tree.numVertices*sizeof(int));
    hipMemcpy(sweptd,  swept, tree.numVertices*sizeof(int), hipMemcpyHostToDevice);

    int* dangling;
    hipMalloc(&dangling, danglingvect.size()*sizeof(int));
    hipMemcpy(dangling, &*danglingvect.begin(), danglingvect.size()*sizeof(int), hipMemcpyHostToDevice);

    hipMemcpyToSymbol(HIP_SYMBOL(trunkStart), &trunkStarter, sizeof(int));

    trunk_augment<<<18,1024>>>(tree.numVertices, danglingvect.size(), valuest, sweptd, dangling);

    hipMemcpy(swept, sweptd, tree.numVertices*sizeof(int), hipMemcpyDeviceToHost);
}

void doCudaStuff(TreeConstructor& tree, std::vector<int*>& minima, std::vector<int*>& saddles, std::vector<int>& counts, int *&swepto, int*& augmentationo, int rounds, int minimaroundsh, int growthroundsh)
{
    int mincount = 0;
    int constexpr zero = 0;

    int*                    neighborsBuffer;
    NeighborsEntry*         neighborsMap;
    int*                    swept;
    int*                    augmentation;
    int*                    minimaQueue;
    unsigned long long int* saddleMap;
    int**                   sweepQueueStarters;

    valuest.resize(dim3(tree.max_x, tree.max_y, tree.max_z));
    valuest.copy(tree.fvalues);

    //Fill device neighbors
    hipMalloc(             &neighborsBuffer,       tree.neighborsBuffer->size()*sizeof(int));
    hipMemcpy(             neighborsBuffer,        &*tree.neighborsBuffer->begin(), tree.neighborsBuffer->size()*sizeof(int), hipMemcpyHostToDevice);
    //Fill device neighborMap
    hipMalloc(             &neighborsMap,          tree.numVertices*sizeof(NeighborsEntry));
    hipMemcpy(             neighborsMap,           &*tree.neighborsMap->begin(), tree.numVertices*sizeof(NeighborsEntry), hipMemcpyHostToDevice);
    //Reserve swept, minimaQueue and saddleMap
    hipMalloc(             &swept,                 tree.numVertices*sizeof(int));
    hipMalloc(             &augmentation,          tree.numVertices*sizeof(int));
    hipMalloc(             &minimaQueue,           tree.numVertices*sizeof(int));
    hipMalloc(             &saddleMap,             tree.numVertices*sizeof(unsigned long long int));
    hipMalloc(             &sweepQueueStarters,    tree.numVertices*sizeof(int*));

    hipMemcpyToSymbol(HIP_SYMBOL(minimaQueueCounter), &mincount, sizeof(int));
    hipMemcpyToSymbol(HIP_SYMBOL(doneCounter), &zero, sizeof(int));
    hipMemcpyToSymbol(HIP_SYMBOL(min_value), &tree.min_val, sizeof(double));
    hipMemcpyToSymbol(HIP_SYMBOL(conversionFactor), &tree.conversionFactor, sizeof(double));
    hipMemcpyToSymbol(HIP_SYMBOL(max_x), &tree.max_x, sizeof(int));
    hipMemcpyToSymbol(HIP_SYMBOL(max_y), &tree.max_y, sizeof(int));
    hipMemcpyToSymbol(HIP_SYMBOL(max_z), &tree.max_z, sizeof(int));
    hipMemcpyToSymbol(HIP_SYMBOL(minimarounds), &minimaroundsh, sizeof(int));
    hipMemcpyToSymbol(HIP_SYMBOL(growthrounds), &growthroundsh, sizeof(int));



    //Start First Round
    std::chrono::high_resolution_clock::time_point start = std::chrono::high_resolution_clock::now();
    std::chrono::duration<double> elapsed_secs;

    //Search for start points
    min_search_preNeigh<<<18,1024>>>(tree.numVertices, valuest, neighborsBuffer, neighborsMap, minimaQueue, swept, augmentation, saddleMap);
    //min_search_<<<18,1024>>>(tree.numVertices, values, minimaQueue, swept, saddleMap);

    //Read number of found start points
    hipMemcpyFromSymbol(&mincount, HIP_SYMBOL(minimaQueueCounter), sizeof(int));

    //Read start points from device
    minima.push_back((int*)malloc(mincount*sizeof(int)));
    hipMemcpy(minima.at(minima.size()-1), minimaQueue, mincount*sizeof(int), hipMemcpyDeviceToHost);
    counts.push_back(mincount);

    //Remember number of start points on device
    hipMemcpyToSymbol(HIP_SYMBOL(minimaCounter), &mincount, sizeof(int));

    //Sweep from all starts
    //min_growth<<<18,1024>>>(tree.numVertices, values, minimaQueue, swept, saddleMap, sweepQueueStarters);
    min_growth_preNeigh<<<18,1024>>>(tree.numVertices, valuest, neighborsBuffer, neighborsMap, minimaQueue, swept, saddleMap, sweepQueueStarters);

    //Clear minimaQueueCounter
    hipMemcpyToSymbol(HIP_SYMBOL(minimaQueueCounter), &zero, sizeof(int));

    //Search for end points
    saddle_search<<<18,1024>>>(tree.numVertices, valuest, neighborsBuffer, neighborsMap, minimaQueue, swept, saddleMap);

    //prepare end points
    get_saddles<<<18,1024>>>(tree.numVertices, valuest, neighborsBuffer, neighborsMap, minimaQueue, swept, saddleMap);

    //Read end points from device
    saddles.push_back((int*)malloc(mincount*sizeof(int)));
    hipMemcpy(saddles.at(saddles.size()-1), minimaQueue, mincount*sizeof(int), hipMemcpyDeviceToHost);

    //Search for start points
    compress_saddles<<<18,1024>>>(tree.numVertices, valuest, neighborsBuffer, neighborsMap, minimaQueue, swept, saddleMap, sweepQueueStarters);

    //Read number of found start points
    hipMemcpyFromSymbol(&mincount, HIP_SYMBOL(minimaQueueCounter), sizeof(int));

    int i = 0;
    while (i < rounds){
        i++;
        //Read start points from device
        minima.push_back((int*)malloc(mincount*sizeof(int)));
        hipMemcpy(minima.at(minima.size()-1), minimaQueue, mincount*sizeof(int), hipMemcpyDeviceToHost);
        counts.push_back(mincount);

        //Remember number of start points on device
        hipMemcpyToSymbol(HIP_SYMBOL(minimaCounter), &mincount, sizeof(int));

        //Prepare sweepQueueStarters
        clean_saddleMap_to_zero<<<18,1024>>>(tree.numVertices, swept, saddleMap);

        size_sweepqueues<<<18,1024>>>(tree.numVertices, valuest, neighborsBuffer, neighborsMap, minimaQueue, swept, saddleMap, sweepQueueStarters);
        create_sweepqueues<<<18,1024>>>(tree.numVertices, valuest, neighborsBuffer, neighborsMap, minimaQueue, swept, saddleMap, sweepQueueStarters);
        fill_sweepqueues<<<18,1024>>>(tree.numVertices, valuest, neighborsBuffer, neighborsMap, minimaQueue, swept, saddleMap, sweepQueueStarters);

        //Compress UF
        compress_UF<<<18,1024>>>(tree.numVertices, valuest, swept, augmentation);

        //hipMemcpy(swepto, swept, tree.numVertices*sizeof(int), hipMemcpyDeviceToHost);
        hipMemcpyToSymbol(HIP_SYMBOL(doneCounter), &zero, sizeof(int));
        //Sweep from all starts
        saddle_growth<<<36,512>>>(tree.numVertices, valuest, neighborsBuffer, neighborsMap, minimaQueue, swept, saddleMap, sweepQueueStarters, i);
        //min_growth<<<18,1024>>>(tree.numVertices, valuest, neighborsBuffer, neighborsMap, minimaQueue, swept);
        //host_min_growth(tree.numVertices, mincount, tree.valuest, &*tree.neighborsBuffer->begin(), &*tree.neighborsMap->begin(), minima.at(i), swepto);

        //Clear minimaQueueCounter
        hipMemcpyToSymbol(HIP_SYMBOL(minimaQueueCounter), &zero, sizeof(int));

        //hipMemcpy(swept, swepto, tree.numVertices*sizeof(int), hipMemcpyHostToDevice);

        //Search for end points
        clean_saddleMap_to_infinity<<<18,1024>>>(tree.numVertices, swept, saddleMap);

        saddle_search<<<18,1024>>>(tree.numVertices, valuest, neighborsBuffer, neighborsMap, minimaQueue, swept, saddleMap);

        //prepare end points
        get_saddles<<<18,1024>>>(tree.numVertices, valuest, neighborsBuffer, neighborsMap, minimaQueue, swept, saddleMap);

        //Read end points from device
        saddles.push_back((int*)malloc(mincount*sizeof(int)));
        hipMemcpy(saddles.at(saddles.size()-1), minimaQueue, mincount*sizeof(int), hipMemcpyDeviceToHost);

        //Search for start points
        compress_saddles<<<18,1024>>>(tree.numVertices, valuest, neighborsBuffer, neighborsMap, minimaQueue, swept, saddleMap, sweepQueueStarters);

        //Read number of found start points
        hipMemcpyFromSymbol(&mincount, HIP_SYMBOL(minimaQueueCounter), sizeof(int));

    }

    minima.push_back((int*)malloc(mincount*sizeof(int)));
    hipMemcpy(minima.at(minima.size()-1), minimaQueue, mincount*sizeof(int), hipMemcpyDeviceToHost);
    counts.push_back(mincount);

    hipMemcpyToSymbol(HIP_SYMBOL(minimaCounter), &mincount, sizeof(int));

    compress_UF<<<18,1024>>>(tree.numVertices, valuest, swept, augmentation);

    flatten_swept<<<18,1024>>>(tree.numVertices, valuest, swept);

    swepto = (int*)malloc(tree.numVertices * sizeof(int));
    hipMemcpy(swepto, swept, tree.numVertices * sizeof(int), hipMemcpyDeviceToHost);

    augmentationo = (int*)malloc(tree.numVertices * sizeof(int));
    hipMemcpy(augmentationo, augmentation, tree.numVertices * sizeof(int), hipMemcpyDeviceToHost);

    hipFree(neighborsBuffer);
    hipFree(neighborsMap);
    hipFree(swept);
    hipFree(augmentation);
    hipFree(minimaQueue);
    hipFree(saddleMap);
    hipFree(sweepQueueStarters);

    elapsed_secs = std::chrono::high_resolution_clock::now() - start;
    //hipMemcpy(swepto, swept, tree.numVertices*sizeof(int), hipMemcpyDeviceToHost);

    std::cout << "Device Time: " << elapsed_secs.count() << std::endl;
}

#endif
